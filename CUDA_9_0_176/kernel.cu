#include "hip/hip_runtime.h"

__device__ int pixel_diff(int a, int b)
{
	return (((16711680 & a) - (16711680 & b)) >> 16) * (((16711680 & a) - (16711680 & b)) >> 16)
		+ (((65280 & a) - (65280 & b)) >> 8) * (((65280 & a) - (65280 & b)) >> 8)
		+ ((255 & a) - (255 & b)) * ((255 & a) - (255 & b));
}

__global__ void kernel(const int* tiles, const int* grid, const int tilecount, const int gridcount, const int tilewidth, int* scores)
{
	int index = blockDim.x * (gridDim.x * blockIdx.y + blockIdx.x) + threadIdx.x;
	int gridindex = index / tilecount;
	int tileindex = index % tilecount;
	if (gridindex < gridcount) {
		int score = 0;
		for (int tilepixel = 0; tilepixel < tilewidth; tilepixel++) {
			score += pixel_diff(
				tiles[tileindex * tilewidth + tilepixel], 
				grid[gridindex * tilewidth + tilepixel]
			);
		}
		scores[index] = score;
	}
	//__syncthreads();
}

int main()
{
	return 0;
}