#include "hip/hip_runtime.h"
#include "math.h"

__device__ int diff(int a, int b)
{
	return (((16711680 & a) - (16711680 & b)) >> 16) * (((16711680 & a) - (16711680 & b)) >> 16)
		+ (((65280 & a) - (65280 & b)) >> 8) * (((65280 & a) - (65280 & b)) >> 8)
		+ ((255 & a) - (255 & b)) * ((255 & a) - (255 & b));
}

__device__ int diff_advanced(int a, int b, const float* weights)
{
	return (int) ((float)(abs((16711680 & a) - (16711680 & b)) >> 16) * weights[0]
		+ (float)abs(((65280 & a) - (65280 & b)) >> 8) * weights[1]
		+ (float)abs((255 & a) - (255 & b)) * weights[2]);
}

//__device__ int diff_advanced(int a, int b, const int* weights)
//{
//	return (((16711680 & a) - (16711680 & b)) >> 16) * (((16711680 & a) - (16711680 & b)) >> 16) * weights[0]
//		+ (((65280 & a) - (65280 & b)) >> 8) * (((65280 & a) - (65280 & b)) >> 8) * weights[1]
//		+ ((255 & a) - (255 & b)) * ((255 & a) - (255 & b)) * weights[2];
//}

__global__ void kernel(const int* tiles, const int* grid, int checks, int tilewidth, int* scores, int* bests, int tileN, int gridN, int count, int top)
{
	//__shared__ int cutoff;
	__shared__ int* best;
	__shared__ int* topscores;

	int block = gridDim.x * blockIdx.y + blockIdx.x;

	if (threadIdx.x == 0) {
		//cutoff = INT_MAX;
		best = new int[top];
		topscores = new int[top];
		for (int i = 0; i < top; i++) {
			topscores[i] = INT_MAX;
		}
	}

	__syncthreads();

	if (block < count) {
		for (int c = 0; c < checks; c++) {
			int t = ((threadIdx.x * checks) + c) * tilewidth;
			if (t < tileN) {
				int g = block * tilewidth;
				if (g < gridN) {
					int score = 0;
					int i = 0;
					while (i < tilewidth && score < topscores[top - 1]) {
						score += diff(tiles[t + i], grid[g + i]);
						i++;
					}
					if (score < topscores[top - 1]) {
						int besttile = (threadIdx.x * checks) + c;;
						for (int i = 0; i < top; i++) {
							if (score < topscores[i]) {
								int temp = topscores[i];
								topscores[i] = score;
								score = temp;
								temp = best[i];
								best[i] = besttile;
								besttile = temp;
							}
						}
					}
				}
			}
		}
	}

	__syncthreads();

	if (threadIdx.x == 0) {
		if (block < count) {
			for (int i = 0; i < top; i++) {
				bests[block * top + i] = best[i];
				scores[block * top + i] = topscores[i];
			}
		}
	}
}

__global__ void kernel_advanced(const int* tiles, const int* grid, const int tilecount, const int gridcount, const int tilewidth, const float* weights, int* scores, int* bests)
{
	int block = gridDim.x * blockIdx.y + blockIdx.x;
	int index = blockDim.x * block + threadIdx.x;
	int gridindex = index / tilecount;
	int tileindex = index % tilecount;
	if (gridindex < gridcount) {
		int score = 0;
		int t = tileindex * tilewidth;
		int g = gridindex * tilewidth;
		for (int i = 0; i < tilewidth; i++) {
			score += diff_advanced(tiles[t + i], grid[g + i], weights);
		}
		scores[index] = score;
		bests[index] = tileindex;
	}
	__syncthreads();

	//if (block < blocks) {
	//	scores[block * threads + threadIdx.x] = INT_MAX;
	//	for (int c = 0; c < checks; c++) {
	//		int t = ((threadIdx.x * checks) + c) * tilewidth;
	//		if (t < tileN) {
	//			int g = block * tilewidth;
	//			if (g < gridN) {
	//				int score = 0;
	//				int i = 0;
	//				while (i < tilewidth && score < scores[block * threads + threadIdx.x]) {
	//					score += diff_advanced(tiles[t + i], grid[g + i], weights);
	//					i++;
	//				}
	//				if (score < scores[block * threads + threadIdx.x]) {
	//					scores[block * threads + threadIdx.x] = score;
	//					bests[block * threads + threadIdx.x] = (threadIdx.x * checks) + c;
	//				}
	//			}
	//		}
	//	}
	//}
}
//__global__ void kernel_advanced(const int* tiles, const int* grid, int checks, int tilewidth, int* scores, int* bests, int tileN, int gridN, int blocks, int dither, const int* weights, const int threads)
//{
//	int block = gridDim.x * blockIdx.y + blockIdx.x;
//	if (block < blocks) {
//		scores[block * threads + threadIdx.x] = INT_MAX;
//		for (int c = 0; c < checks; c++) {
//			int t = ((threadIdx.x * checks) + c) * tilewidth;
//			if (t < tileN) {
//				int g = block * tilewidth;
//				if (g < gridN) {
//					int score = 0;
//					int i = 0;
//					while (i < tilewidth && score < scores[block * threads + threadIdx.x]) {
//						score += diff_advanced(tiles[t + i], grid[g + i], weights);
//						i++;
//					}
//					if (score < scores[block * threads + threadIdx.x]) {
//						scores[block * threads + threadIdx.x] = score;
//						bests[block * threads + threadIdx.x] = (threadIdx.x * checks) + c;
//					}
//				}
//			}
//		}
//	}
//}

int main()
{
	return 0;
}